
#include <hip/hip_runtime.h>
#include <cstdio>

__global__
void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < n; i += gridDim.x * blockDim.x) {
        y[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 1 << 20;

    float *x, *y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int blockNum = (N - blockSize + 1) / blockSize;
    add<<<blockNum, blockSize>>>(N, x, y);

    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) {
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    }
    if (errAsync != hipSuccess) {
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    }

    hipDeviceSynchronize();

    hipFree(x);
    hipFree(y);
}