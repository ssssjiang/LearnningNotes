
#include <hip/hip_runtime.h>
#include <cstring>
#include <cstdio>

class Managed
{
public:
    void *operator new(size_t len)
    {
        void *ptr;
        hipMallocManaged(&ptr, len);
        hipDeviceSynchronize();
        return ptr;
    }

    void operator delete(void *ptr)
    {
        hipDeviceSynchronize();
        hipFree(ptr);
    }
};

struct DataElement : public Managed
{
    char *name;
    int value;
};

__global__
void Kernel(DataElement *elem)
{
    printf("On device: name=%s, value=%d\n", elem->name, elem->value);

    elem->name[0] = 'd';
    elem->value++;
}

void launch(DataElement *elem) {
    Kernel<<<1, 1>>>(elem);
    hipDeviceSynchronize();
}

int main()
{
    DataElement *e = new DataElement;

    e->value = 10;
    hipMallocManaged((void**)&(e->name), sizeof(char) * (strlen("hello") + 1));
    strcpy(e->name, "hello");

    launch(e);

    printf("On host: name=%s, value=%d\n", e->name, e->value);

    hipFree(e->name);
    delete e;

    hipDeviceReset();
}