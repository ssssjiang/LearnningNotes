
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstring>

// Managed Base Class - inherit from this to automatically
// allocate objects in Unified Memory
class Managed
{
public:
    void *operator new(size_t len)
    {
        void *ptr;
        hipMallocManaged(&ptr, len);
        hipDeviceSynchronize();
        return ptr;
    }

    void operator delete(void *ptr)
    {
        hipDeviceSynchronize();
        hipFree(ptr);
    }
};

// String Class for Managed Memory
class String : public Managed
{
    int length;
    char *data;

public:
    String() : length(0), data(0) {}

    // Constructor for C-string initilizer
    String(const char *s) : length(0), data(0) 
    {
        _realloc(strlen(s));
        strcpy(data, s);
    }

    // Copy constructor
    String(const String& s) : length(0), data(0)
    {
        _realloc(s.length);
        strcpy(data, s.data);
    }

    ~String() {hipFree(data); }

    // Assignment operator
    String& operator=(const char *s)
    {
        _realloc(strlen(s));
        strcpy(data, s);
        return *this;
    }

    // Element access (from host or device)
    __host__ __device__
    char& operator[](int pos) { return data[pos]; }

    // C-string access
    __host__ __device__
    const char* c_str() const { return data; }

private:
    void _realloc(int len)
    {
        hipFree(data);
        length = len;
        hipMallocManaged(&data, length + 1);
    }
};

struct DataElement : public Managed
{
    String name;
    int value;
};

__global__
void Kernel_by_pointer(DataElement *elem)
{
    printf("On device by pointer: name=%s, value=%d\n", elem->name.c_str(), elem->value);

    elem->name[0] = 'p';
    elem->value++;
}

__global__
void Kernel_by_ref(DataElement &elem)
{
    printf("On device by ref: name=%s, value=%d\n", elem.name.c_str(), elem.value);

    elem.name[0] = 'r';
    elem.value++;
}

__global__
void Kernel_by_value(DataElement elem)
{
    printf("On device by value: name=%s, value=%d\n", elem.name.c_str(), elem.value);

    elem.name[0] = 'v';
    elem.value++;
}

void launch_by_pointer(DataElement *elem)
{
    Kernel_by_pointer<<<1, 1>>>(elem);
    hipDeviceSynchronize();
}

void launch_by_ref(DataElement &elem)
{
    Kernel_by_ref<<<1, 1>>>(elem);
    hipDeviceSynchronize();
}

void launch_by_value(DataElement elem)
{
    Kernel_by_value<<<1, 1>>>(elem);
    hipDeviceSynchronize();
}

int main()
{
    DataElement *e = new DataElement();

    e->value = 10;
    e->name = "hello";

    launch_by_pointer(e);

    printf("on host (after by-pointer): name=%s, value=%d\n", e->name.c_str(), e->value);

    launch_by_ref(*e);

    printf("On host (after by-ref): name=%s, value=%d\n", e->name.c_str(), e->value);

    launch_by_value(*e);

    printf("On host (after by-value): name=%s, value=%d\n", e->name.c_str(), e->value);

    hipDeviceReset();
}